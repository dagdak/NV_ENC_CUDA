
#include <hip/hip_runtime.h>

// BlockDim = 16x16
//GridDim = w/16*h/16
extern "C" __global__ void RGB2NV12( unsigned char *in_rgb, unsigned char *nv12,
        int rgb_width, int rgb_height, int rgb_pitch, int nv12_pitch )
{
    unsigned char *rgb1,*rgb2,*rgb3,*rgb4, r,g,b,u,v;
    unsigned char *pYDst, *pUVDst;
    int x,y,uv_y;
    x = blockIdx.x*blockDim.x+threadIdx.x;
    y = blockIdx.y*blockDim.y+threadIdx.y;//Y 
    uv_y = y + (rgb_height<<1);// UV 

    if ((x < rgb_width) && (y < rgb_height))
    {
        rgb1 = in_rgb + (y << 1) * rgb_pitch + (x << 1) * 3;
        rgb2 = in_rgb + (y << 1) * rgb_pitch + ((x << 1) + 1) * 3;
        rgb3 = rgb1 + rgb_pitch;
        rgb4 = rgb2 + rgb_pitch;

        pYDst = nv12 + (y << 1)*nv12_pitch;
        u = -0.09991f * rgb1[0] - 0.33609f * rgb1[1] + 0.436f * rgb1[2] + 128;
        v = 0.614f * rgb1[0] - 0.55861f * rgb1[1] - 0.05639f * rgb1[2] + 128;
        pYDst[x << 1] = (rgb1[0]+ rgb1[1] + rgb1[2])/3.492f + 104.339f - 0.446f*u - 0.224f*v;//Y 

        u = -0.09991f * rgb2[0] - 0.33609f * rgb2[1] + 0.436f * rgb2[2] + 128;
        v = 0.614f*rgb2[0]  - 0.55861f*rgb2[1]  - 0.05639f*rgb2[2]  + 128;
        pYDst[(x << 1) + 1] = (rgb2[0] + rgb2[1] +rgb2[2])/3.492f + 104.339f - 0.446f*u - 0.224f*v;//Y

        pYDst = nv12 + ((y << 1) + 1)*nv12_pitch;
        u = -0.09991f * rgb3[0]- 0.33609f * rgb3[1] + 0.436f * rgb3[2] + 128;
        v = 0.614f * rgb3[0]  - 0.55861f * rgb3[1] - 0.05639f * rgb3[2] + 128;
        pYDst[x << 1] = (rgb3[0]+ rgb3[1] + rgb3[2])/3.492f + 104.339f - 0.446f*u - 0.224f*v;//Y

        u = -0.09991f * rgb4[0] - 0.33609f * rgb4[1] + 0.436f * rgb4[2] + 128;
        v = 0.614f * rgb4[0] - 0.55861f * rgb4[1] - 0.05639f * rgb4[2] + 128;
        pYDst[(x << 1) + 1] = (rgb4[0]+ rgb4[1] + rgb4[2])/3.492f + 104.339f - 0.446f*u - 0.224f*v;//Y

        r = (rgb1[0] + rgb2[0] + rgb3[0] + rgb4[0])/4;
        g = (rgb1[1] + rgb2[1] + rgb3[1] + rgb4[1])/4;
        b = (rgb1[2] + rgb2[2] + rgb3[2] + rgb4[2])/4;
        pUVDst = nv12 + uv_y*nv12_pitch;
        pUVDst[x << 1] =  -0.09991f*r - 0.33609f*g + 0.436f*b + 128;//U
        pUVDst[(x << 1) + 1] = 0.614f*r - 0.55861f*g - 0.05639f*b + 128;//V

    }
}

